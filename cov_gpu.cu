#include "hip/hip_runtime.h"
#include "cov_gpu.hpp"
#include <stdio.h>

__device__ REAL cov_val_d(int n_dim, REAL *x, REAL *y, REAL *hypers)
{
	REAL scale = hypers[n_dim];
	REAL s = 0.0;
	for (unsigned i=0; i<n_dim; i++)
	{
		REAL d = (x[i]-y[i])/hypers[i];
		s += d * d;
	}
	return scale * exp(-0.5*s);
}

__global__ void cov_val_d_wrapper(REAL *result_d, int n_dim, REAL *x, REAL *y, REAL *hypers)
{
	*result_d = cov_val_d(n_dim,x,y,hypers);
}

void cov_val_wrapper(REAL *result_d, int n_dim, REAL *x, REAL *y, REAL *hypers)
{
	cov_val_d_wrapper<<<1,1>>>(result_d, n_dim, x, y, hypers);
}

// Computes the vector of covariances with a new point with (the vector 'k' in the notation I have been using)
// could use thrust device vectors or similar
// just values for now -- fix when working
__global__ void cov_all_kernel(REAL *result, int N, int n_dim, REAL *xnew, REAL *xs, REAL *theta)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<N) {
		result[i] = cov_val_d(n_dim, xnew, xs + n_dim*i, theta);
	}
}

__global__ void cov_batch(REAL *result, int Nnew, int N, int n_dim, REAL *xsnew, 
			  REAL *xs, REAL *theta)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i<N && j<Nnew) {
		result[j+Nnew*i] = cov_val_d(n_dim, xsnew + n_dim*j, xs + n_dim*i, theta);
	}
}

// wrapper
void cov_all_wrapper(REAL *result, int N, int n_dim, REAL *xnew, REAL *xs, REAL *theta)
{
	const int threads_per_block = 256;
	cov_all_kernel <<< 10, threads_per_block >>> (result, N, n_dim, xnew, xs, theta);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}

void cov_batch_wrapper(REAL *result, int Nnew, int N, int n_dim, REAL *xsnew, 
			  REAL *xs, REAL *theta)
{
	dim3 threads_per_block(8,32);
	dim3 blocks(250,625);
	cov_batch <<< blocks, threads_per_block >>> (result, Nnew, N, n_dim, xsnew, xs, theta);

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
}
